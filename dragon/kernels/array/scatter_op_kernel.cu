
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/math_functions.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernels {

namespace {

template <typename T, int D>
__global__ void _ScatterElements(
    const int N,
    const int axis,
    const int num_dims,
    const T value,
    const SimpleArray<int, D> X_dims,
    const SimpleArray<int, D> Y_strides,
    const int64_t* index,
    T* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    int yi = 0, tmp = i;
    for (int d = num_dims - 1; d >= 0; --d) {
      int r;
      FIXED_DIVISOR_DIV_MOD(X_dims.data[d], tmp, &tmp, &r);
      yi += (d == axis ? index[i] : r) * Y_strides.data[d];
    }
    y[yi] = value;
  }
}

template <typename T, int D>
__global__ void _ScatterElements(
    const int N,
    const int axis,
    const int num_dims,
    const SimpleArray<int, D> X_dims,
    const SimpleArray<int, D> X_strides,
    const SimpleArray<int, D> Y_strides,
    const int64_t* index,
    const T* x,
    T* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    int xi = 0, yi = 0, tmp = i;
    for (int d = num_dims - 1; d >= 0; --d) {
      int r;
      FIXED_DIVISOR_DIV_MOD(X_dims.data[d], tmp, &tmp, &r);
      xi += r * X_strides.data[d];
      yi += (d == axis ? index[i] : r) * Y_strides.data[d];
    }
    y[yi] = x[xi];
  }
}

template <typename T, typename AccT, int D>
__global__ void _ScatterAdd(
    const int N,
    const int axis,
    const int num_dims,
    const SimpleArray<int, D> X_dims,
    const SimpleArray<int, D> X_strides,
    const SimpleArray<int, D> Y_strides,
    const int64_t* index,
    const T* x,
    AccT* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    int xi = 0, yi = 0, tmp = i;
    for (int d = num_dims - 1; d >= 0; --d) {
      int r;
      FIXED_DIVISOR_DIV_MOD(X_dims.data[d], tmp, &tmp, &r);
      xi += r * X_strides.data[d];
      yi += (d == axis ? index[i] : r) * Y_strides.data[d];
    }
    math::utils::AtomicAdd(y + yi, convert::To<AccT>(x[xi]));
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

#define DEFINE_KERNEL_LAUNCHER(name, T)                                        \
  template <>                                                                  \
  void name<T, CUDAContext>(                                                   \
      const int axis,                                                          \
      const int num_dims,                                                      \
      const T value,                                                           \
      const int64_t* dims,                                                     \
      const int64_t* y_strides,                                                \
      const int64_t* index,                                                    \
      T* y,                                                                    \
      CUDAContext* ctx) {                                                      \
    CUDA_TENSOR_DIMS_CHECK(num_dims);                                          \
    SimpleArray<int, CUDA_TENSOR_MAX_DIMS> X_dims;                             \
    SimpleArray<int, CUDA_TENSOR_MAX_DIMS> Y_strides;                          \
    const auto N =                                                             \
        std::accumulate(dims, dims + num_dims, 1, std::multiplies<int64_t>()); \
    for (int i = 0; i < num_dims; ++i) {                                       \
      X_dims.data[i] = dims[i];                                                \
      Y_strides.data[i] = y_strides[i];                                        \
    }                                                                          \
    _##name<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->cuda_stream()>>>(          \
        N, axis, num_dims, value, X_dims, Y_strides, index, y);                \
  }

DEFINE_KERNEL_LAUNCHER(ScatterElements, bool);
DEFINE_KERNEL_LAUNCHER(ScatterElements, uint8_t);
DEFINE_KERNEL_LAUNCHER(ScatterElements, int8_t);
DEFINE_KERNEL_LAUNCHER(ScatterElements, int);
DEFINE_KERNEL_LAUNCHER(ScatterElements, int64_t);
DEFINE_KERNEL_LAUNCHER(ScatterElements, float16);
DEFINE_KERNEL_LAUNCHER(ScatterElements, float);
DEFINE_KERNEL_LAUNCHER(ScatterElements, double);
#undef DEFINE_KERNEL_LAUNCHER

#define DEFINE_KERNEL_LAUNCHER(name, T)                                        \
  template <>                                                                  \
  void name<T, CUDAContext>(                                                   \
      const int axis,                                                          \
      const int num_dims,                                                      \
      const int64_t* dims,                                                     \
      const int64_t* x_strides,                                                \
      const int64_t* y_strides,                                                \
      const int64_t* index,                                                    \
      const T* x,                                                              \
      T* y,                                                                    \
      CUDAContext* ctx) {                                                      \
    CUDA_TENSOR_DIMS_CHECK(num_dims);                                          \
    SimpleArray<int, CUDA_TENSOR_MAX_DIMS> X_dims;                             \
    SimpleArray<int, CUDA_TENSOR_MAX_DIMS> X_strides;                          \
    SimpleArray<int, CUDA_TENSOR_MAX_DIMS> Y_strides;                          \
    const auto N =                                                             \
        std::accumulate(dims, dims + num_dims, 1, std::multiplies<int64_t>()); \
    for (int i = 0; i < num_dims; ++i) {                                       \
      X_dims.data[i] = dims[i];                                                \
      X_strides.data[i] = x_strides[i];                                        \
      Y_strides.data[i] = y_strides[i];                                        \
    }                                                                          \
    _##name<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->cuda_stream()>>>(          \
        N, axis, num_dims, X_dims, X_strides, Y_strides, index, x, y);         \
  }

DEFINE_KERNEL_LAUNCHER(ScatterElements, bool);
DEFINE_KERNEL_LAUNCHER(ScatterElements, uint8_t);
DEFINE_KERNEL_LAUNCHER(ScatterElements, int8_t);
DEFINE_KERNEL_LAUNCHER(ScatterElements, int);
DEFINE_KERNEL_LAUNCHER(ScatterElements, int64_t);
DEFINE_KERNEL_LAUNCHER(ScatterElements, float16);
DEFINE_KERNEL_LAUNCHER(ScatterElements, float);
DEFINE_KERNEL_LAUNCHER(ScatterElements, double);
#undef DEFINE_KERNEL_LAUNCHER

#define DEFINE_KERNEL_LAUNCHER(name, T, AccT)                                  \
  template <>                                                                  \
  void name<T, AccT, CUDAContext>(                                             \
      const int axis,                                                          \
      const int num_dims,                                                      \
      const int64_t* dims,                                                     \
      const int64_t* x_strides,                                                \
      const int64_t* y_strides,                                                \
      const int64_t* index,                                                    \
      const T* x,                                                              \
      AccT* y,                                                                 \
      CUDAContext* ctx) {                                                      \
    CUDA_TENSOR_DIMS_CHECK(num_dims);                                          \
    SimpleArray<int, CUDA_TENSOR_MAX_DIMS> X_dims;                             \
    SimpleArray<int, CUDA_TENSOR_MAX_DIMS> X_strides;                          \
    SimpleArray<int, CUDA_TENSOR_MAX_DIMS> Y_strides;                          \
    const auto N =                                                             \
        std::accumulate(dims, dims + num_dims, 1, std::multiplies<int64_t>()); \
    for (int i = 0; i < num_dims; ++i) {                                       \
      X_dims.data[i] = dims[i];                                                \
      X_strides.data[i] = x_strides[i];                                        \
      Y_strides.data[i] = y_strides[i];                                        \
    }                                                                          \
    _##name<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->cuda_stream()>>>(          \
        N,                                                                     \
        axis,                                                                  \
        num_dims,                                                              \
        X_dims,                                                                \
        X_strides,                                                             \
        Y_strides,                                                             \
        index,                                                                 \
        reinterpret_cast<const math::ScalarType<T>::type*>(x),                 \
        y);                                                                    \
  }

DEFINE_KERNEL_LAUNCHER(ScatterAdd, uint8_t, uint8_t);
DEFINE_KERNEL_LAUNCHER(ScatterAdd, int8_t, int8_t);
DEFINE_KERNEL_LAUNCHER(ScatterAdd, int, int)
DEFINE_KERNEL_LAUNCHER(ScatterAdd, int64_t, int64_t)
DEFINE_KERNEL_LAUNCHER(ScatterAdd, float16, float);
DEFINE_KERNEL_LAUNCHER(ScatterAdd, float, float)
DEFINE_KERNEL_LAUNCHER(ScatterAdd, double, float);
#undef DEFINE_KERNEL_LAUNCHER

} // namespace kernels

} // namespace dragon

#endif // USE_CUDA
