
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernel {

namespace {

template <typename T>
__global__ void _IndexSelect(
    const int nthreads,
    const int inner_dim,
    const int axis_dim,
    const int num_indices,
    const int64_t* indices,
    const T* x,
    T* y) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    const int j = yi % inner_dim;
    const int i = yi / inner_dim / num_indices;
#if __CUDA_ARCH__ >= 350
    int index = __ldg(indices + ((yi / inner_dim) % num_indices));
#else
    int index = indices[(yi / inner_dim) % num_indices];
#endif
    index = index >= 0 ? index : index + axis_dim;
    y[yi] = x[(i * axis_dim + index) * inner_dim + j];
  }
}

template <typename T>
__global__ void _IndexSelectGrad(
    const int nthreads,
    const int inner_dim,
    const int axis_dim,
    const int num_indices,
    const int64_t* indices,
    const T* dy,
    T* dx) {
  CUDA_1D_KERNEL_LOOP(ti, nthreads) {
    const int i = ti / inner_dim;
    const int j = ti % inner_dim;
    const int c = i * axis_dim * inner_dim + j;
    const T* offset_dy = dy + i * num_indices * inner_dim + j;
    for (int k = 0; k < num_indices; ++k) {
#if __CUDA_ARCH__ >= 350
      int index = __ldg(indices + k);
#else
      int index = indices[k];
#endif
      index = index >= 0 ? index : index + axis_dim;
      dx[c + index * inner_dim] += (*offset_dy);
      offset_dy += inner_dim;
    }
  }
}

template <>
__global__ void _IndexSelectGrad<half>(
    const int nthreads,
    const int inner_dim,
    const int axis_dim,
    const int num_indices,
    const int64_t* indices,
    const half* dy,
    half* dx) {
  CUDA_1D_KERNEL_LOOP(ti, nthreads) {
#if __CUDA_ARCH__ >= 530
    const int i = ti / inner_dim;
    const int j = ti % inner_dim;
    const int c = i * axis_dim * inner_dim + j;
    const half* offset_dy = dy + i * num_indices * inner_dim + j;
    for (int k = 0; k < num_indices; ++k) {
      int index = __ldg(indices + j);
      index = index >= 0 ? index : index + axis_dim;
      index = c + index * inner_dim;
      dx[index] = __hadd(dx[index], *(offset_dy));
      offset_dy += inner_dim;
    }
#endif
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

template <>
void IndexSelectGrad<float16, CUDAContext>(
    const int outer_dim,
    const int inner_dim,
    const int axis_dim,
    const int num_indices,
    const int64_t* indices,
    const float16* dy,
    float16* dx,
    CUDAContext* ctx) {
  const int nthreads = outer_dim * inner_dim;
  _IndexSelectGrad<<<
      CUDA_BLOCKS(nthreads),
      CUDA_THREADS,
      0,
      ctx->cuda_stream()>>>(
      nthreads,
      inner_dim,
      axis_dim,
      num_indices,
      indices,
      reinterpret_cast<const half*>(dy),
      reinterpret_cast<half*>(dx));
} // IndexSelectGrad

#define DEFINE_KERNEL_LAUNCHER(T)                                   \
  template <>                                                       \
  void IndexSelect<T, CUDAContext>(                                 \
      const int outer_dim,                                          \
      const int inner_dim,                                          \
      const int axis_dim,                                           \
      const int num_indices,                                        \
      const int64_t* indices,                                       \
      const T* x,                                                   \
      T* y,                                                         \
      CUDAContext* ctx) {                                           \
    const int nthreads = outer_dim * num_indices * inner_dim;       \
    _IndexSelect<<<                                                 \
        CUDA_BLOCKS(nthreads),                                      \
        CUDA_THREADS,                                               \
        0,                                                          \
        ctx->cuda_stream()>>>(                                      \
        nthreads, inner_dim, axis_dim, num_indices, indices, x, y); \
  }

#define DEFINE_GRAD_KERNEL_LAUNCHER(T)                                \
  template <>                                                         \
  void IndexSelectGrad<T, CUDAContext>(                               \
      const int outer_dim,                                            \
      const int inner_dim,                                            \
      const int axis_dim,                                             \
      const int num_indices,                                          \
      const int64_t* indices,                                         \
      const T* dy,                                                    \
      T* dx,                                                          \
      CUDAContext* ctx) {                                             \
    const int nthreads = outer_dim * inner_dim;                       \
    _IndexSelectGrad<<<                                               \
        CUDA_BLOCKS(nthreads),                                        \
        CUDA_THREADS,                                                 \
        0,                                                            \
        ctx->cuda_stream()>>>(                                        \
        nthreads, inner_dim, axis_dim, num_indices, indices, dy, dx); \
  }

DEFINE_KERNEL_LAUNCHER(bool);
DEFINE_KERNEL_LAUNCHER(int8_t);
DEFINE_KERNEL_LAUNCHER(uint8_t);
DEFINE_KERNEL_LAUNCHER(int);
DEFINE_KERNEL_LAUNCHER(int64_t);
DEFINE_KERNEL_LAUNCHER(float16);
DEFINE_KERNEL_LAUNCHER(float);
DEFINE_KERNEL_LAUNCHER(double);

DEFINE_GRAD_KERNEL_LAUNCHER(float);
DEFINE_GRAD_KERNEL_LAUNCHER(double);

#undef DEFINE_KERNEL_LAUNCHER
#undef DEFINE_GRAD_KERNEL_LAUNCHER

} // namespace kernel

} // namespace dragon

#endif // USE_CUDA