
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/cast.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernel {

namespace {

template <typename T>
__global__ void
_Clip(const int nthreads, const T low, const T high, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    y[i] = max(low, min(x[i], high));
  }
}

template <>
__global__ void _Clip<half>(
    const int nthreads,
    const half low,
    const half high,
    const half* x,
    half* y) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 530
    const half val = __hlt(__ldg(x + i), high) ? __ldg(x + i) : high;
    y[i] = __hgt(val, low) ? val : low;
#endif
  }
}

template <typename T>
__global__ void _ClipGrad(
    const int nthreads,
    const T low,
    const T high,
    const T* dy,
    const T* x,
    T* dx) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 350
    dx[i] = __ldg(x + i) < low || __ldg(x + i) > high ? T(0) : dy[i];
#else
    dx[i] = x[i] < low || x[i] > high ? T(0) : dy[i];
#endif
  }
}

template <>
__global__ void _ClipGrad<half>(
    const int nthreads,
    const half low,
    const half high,
    const half* dy,
    const half* x,
    half* dx) {
  const half kZero = __float2half(0.f);
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 530
    dx[i] =
        __hlt(__ldg(x + i), low) || __hgt(__ldg(x + i), high) ? kZero : dy[i];
#endif
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

template <>
void Clip<float16, CUDAContext>(
    const int count,
    const float low,
    const float high,
    const float16* x,
    float16* y,
    CUDAContext* ctx) {
  _Clip<<<CUDA_BLOCKS(count), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
      count,
      cast::to<half>(low),
      cast::to<half>(high),
      reinterpret_cast<const half*>(x),
      reinterpret_cast<half*>(y));
}

template <>
void ClipGrad<float16, CUDAContext>(
    const int count,
    const float low,
    const float high,
    const float16* dy,
    const float16* x,
    float16* dx,
    CUDAContext* ctx) {
  _ClipGrad<<<CUDA_BLOCKS(count), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
      count,
      cast::to<half>(low),
      cast::to<half>(high),
      reinterpret_cast<const half*>(dy),
      reinterpret_cast<const half*>(x),
      reinterpret_cast<half*>(dx));
} // ClipGrad

#define DEFINE_KERNEL_LAUNCHER(T)                                       \
  template <>                                                           \
  void Clip<T, CUDAContext>(                                            \
      const int count,                                                  \
      const float low,                                                  \
      const float high,                                                 \
      const T* x,                                                       \
      T* y,                                                             \
      CUDAContext* ctx) {                                               \
    _Clip<<<CUDA_BLOCKS(count), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
        count, cast::to<T>(low), cast::to<T>(high), x, y);              \
  }

#define DEFINE_GRAD_KERNEL_LAUNCHER(T)                                      \
  template <>                                                               \
  void ClipGrad<T, CUDAContext>(                                            \
      const int count,                                                      \
      const float low,                                                      \
      const float high,                                                     \
      const T* dy,                                                          \
      const T* x,                                                           \
      T* dx,                                                                \
      CUDAContext* ctx) {                                                   \
    _ClipGrad<<<CUDA_BLOCKS(count), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
        count, cast::to<T>(low), cast::to<T>(high), dy, x, dx);             \
  }

DEFINE_KERNEL_LAUNCHER(int8_t);
DEFINE_KERNEL_LAUNCHER(uint8_t);
DEFINE_KERNEL_LAUNCHER(int);
DEFINE_KERNEL_LAUNCHER(int64_t);
DEFINE_KERNEL_LAUNCHER(float);
DEFINE_KERNEL_LAUNCHER(double);

DEFINE_GRAD_KERNEL_LAUNCHER(float);
DEFINE_GRAD_KERNEL_LAUNCHER(double);

#undef DEFINE_KERNEL_LAUNCHER
#undef DEFINE_GRAD_KERNEL_LAUNCHER

} // namespace kernel

} // namespace dragon

#endif // USE_CUDA