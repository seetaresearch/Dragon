
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernel {

namespace {

template <typename LogitType, typename TargetType>
__global__ void _NLLLoss(
    const int nthreads,
    const int axis_dim,
    const int inner_dim,
    const int ignore_index,
    const LogitType* log_prob,
    const TargetType* target,
    LogitType* loss,
    int* mask) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    const int i = yi / inner_dim;
    const int j = yi % inner_dim;
    const int label = target[i * inner_dim + j];
    if (label == ignore_index) {
      loss[yi] = mask[yi] = 0;
    } else {
      loss[yi] = -log_prob[(i * axis_dim + label) * inner_dim + j];
      mask[yi] = 1;
    }
  }
}

template <typename LogitType, typename TargetType>
__global__ void _NLLLossGrad(
    const int nthreads,
    const int axis_dim,
    const int inner_dim,
    const int ignore_index,
    const LogitType* log_prob,
    const TargetType* target,
    LogitType* dx,
    int* mask) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    const int i = yi / inner_dim;
    const int j = yi % inner_dim;
    const int label = target[i * inner_dim + j];
    if (label == ignore_index) {
      mask[yi] = 0;
    } else {
      dx[(i * axis_dim + label) * inner_dim + j] = LogitType(-1);
      mask[yi] = 1;
    }
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

#define DEFINE_KERNEL_LAUNCHER(name, LogitType, TargetType)                  \
  template <>                                                                \
  void name<LogitType, TargetType, CUDAContext>(                             \
      const int outer_dim,                                                   \
      const int axis_dim,                                                    \
      const int inner_dim,                                                   \
      const int ignore_index,                                                \
      const LogitType* log_prob,                                             \
      const TargetType* target,                                              \
      LogitType* loss,                                                       \
      int* mask,                                                             \
      CUDAContext* ctx) {                                                    \
    auto nthreads = outer_dim * inner_dim;                                   \
    _##name<<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
        nthreads,                                                            \
        axis_dim,                                                            \
        inner_dim,                                                           \
        ignore_index,                                                        \
        log_prob,                                                            \
        target,                                                              \
        loss,                                                                \
        mask);                                                               \
  }

DEFINE_KERNEL_LAUNCHER(NLLLoss, float, float);
DEFINE_KERNEL_LAUNCHER(NLLLoss, float, int64_t);
DEFINE_KERNEL_LAUNCHER(NLLLoss, double, double);
DEFINE_KERNEL_LAUNCHER(NLLLoss, double, int64_t);

DEFINE_KERNEL_LAUNCHER(NLLLossGrad, float, float);
DEFINE_KERNEL_LAUNCHER(NLLLossGrad, float, int64_t);
DEFINE_KERNEL_LAUNCHER(NLLLossGrad, double, double);
DEFINE_KERNEL_LAUNCHER(NLLLossGrad, double, int64_t);

#undef DEFINE_KERNEL_LAUNCHER

} // namespace kernel

} // namespace dragon

#endif // USE_CUDA