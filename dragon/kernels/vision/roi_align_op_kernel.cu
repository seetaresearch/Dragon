
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/math_functions.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernels {

namespace {

template <typename T>
__device__ float
_RoiAlignIntp(const int H, const int W, float h, float w, const T* x) {
  if (h < -1.f || h > H || w < -1.f || w > W) return T(0);

  if (h <= 0.f) h = 0.f;
  if (w <= 0.f) w = 0.f;

  int ti = (int)h, bi;
  int li = (int)w, ri;

  if (ti < H - 1) {
    bi = ti + 1;
  } else {
    ti = bi = H - 1;
    h = (float)ti;
  }

  if (li < W - 1) {
    ri = li + 1;
  } else {
    ri = li = W - 1;
    w = (float)li;
  }

  const float tl = convert::To<float>(__ldg(x + ti * W + li));
  const float tr = convert::To<float>(__ldg(x + ti * W + ri));
  const float bl = convert::To<float>(__ldg(x + bi * W + li));
  const float br = convert::To<float>(__ldg(x + bi * W + ri));

  const float v = h - ti;
  const float u = w - li;
  const float t = tl + (tr - tl) * u;
  const float b = bl + (br - bl) * u;

  return t + (b - t) * v;
}

template <typename T>
__device__ void _RoiAlignIntpParam(
    const int H,
    const int W,
    float h,
    float w,
    int& ti,
    int& bi,
    int& li,
    int& ri,
    T& v,
    T& u) {
  if (h < -1.f || h > H || w < -1.f || w > W) {
    li = ri = ti = bi = -1;
    return;
  }

  if (h <= 0.f) h = 0.f;
  if (w <= 0) w = 0.f;

  ti = (int)h;
  li = (int)w;

  if (ti < H - 1) {
    bi = ti + 1;
  } else {
    ti = bi = H - 1;
    h = (float)ti;
  }

  if (li < W - 1) {
    ri = li + 1;
  } else {
    ri = li = W - 1;
    w = (float)li;
  }

  v = h - ti;
  u = w - li;
}

template <typename T, typename AccT>
__global__ void _RoiAlign(
    const int nthreads,
    const int C,
    const int H,
    const int W,
    const int out_h,
    const int out_w,
    const float spatial_scale,
    const int sampling_ratio,
    const T* x,
    const float* rois,
    T* y) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    const int w_out = yi % out_w;
    const int h_out = (yi / out_w) % out_h;
    const int c = (yi / out_w / out_h) % C;
    const int n = yi / out_w / out_h / C;

    const float* roi = rois + n * 5;
    const int batch_ind = roi[0];

    if (batch_ind < 0) {
      y[yi] = convert::To<T>(0.f);
      continue;
    }

    const float roi_wstart = roi[1] * spatial_scale;
    const float roi_hstart = roi[2] * spatial_scale;
    const float roi_wend = roi[3] * spatial_scale;
    const float roi_hend = roi[4] * spatial_scale;

    const float roi_w = max(roi_wend - roi_wstart, 1.f);
    const float roi_h = max(roi_hend - roi_hstart, 1.f);
    const float bin_h = roi_h / float(out_h);
    const float bin_w = roi_w / float(out_w);

    const float hstart = roi_hstart + h_out * bin_h;
    const float wstart = roi_wstart + w_out * bin_w;

    const int grid_h =
        sampling_ratio > 0 ? sampling_ratio : int(ceil(roi_h / float(out_h)));
    const int grid_w =
        sampling_ratio > 0 ? sampling_ratio : int(ceil(roi_w / float(out_w)));

    const T* offset_x = x + (batch_ind * C + c) * H * W;
    AccT val = AccT(0);
    for (int i = 0; i < grid_h; i++) {
      const float h = hstart + (i + .5f) * bin_h / grid_h;
      for (int j = 0; j < grid_w; j++) {
        const float w = wstart + (j + .5f) * bin_w / grid_w;
        val += _RoiAlignIntp(H, W, h, w, offset_x);
      }
    }
    y[yi] = convert::To<T>(val / AccT(grid_h * grid_w));
  }
}

template <typename T, typename AccT>
__global__ void _RoiAlignGrad(
    const int nthreads,
    const int C,
    const int H,
    const int W,
    const int out_h,
    const int out_w,
    const float spatial_scale,
    const int sampling_ratio,
    const T* dy,
    const float* rois,
    AccT* dx) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    const int w_out = yi % out_w;
    const int h_out = (yi / out_w) % out_h;
    const int c = (yi / out_w / out_h) % C;
    const int n = yi / out_w / out_h / C;

    const float* roi = rois + n * 5;
    const int batch_ind = roi[0];

    if (batch_ind < 0) continue;

    const float roi_wstart = roi[1] * spatial_scale;
    const float roi_hstart = roi[2] * spatial_scale;
    const float roi_wend = roi[3] * spatial_scale;
    const float roi_hend = roi[4] * spatial_scale;

    const float roi_w = max(roi_wend - roi_wstart, 1.f);
    const float roi_h = max(roi_hend - roi_hstart, 1.f);
    const float bin_h = roi_h / float(out_h);
    const float bin_w = roi_w / float(out_w);

    const float hstart = roi_hstart + h_out * bin_h;
    const float wstart = roi_wstart + w_out * bin_w;

    const int grid_h =
        sampling_ratio > 0 ? sampling_ratio : int(ceil(roi_h / float(out_h)));
    const int grid_w =
        sampling_ratio > 0 ? sampling_ratio : int(ceil(roi_w / float(out_w)));
    const float dyi = convert::To<float>(dy[yi]) / float(grid_h * grid_w);
    float* offset_dx = dx + (batch_ind * C + c) * H * W;

    for (int i = 0; i < grid_h; i++) {
      const float h = hstart + (i + .5f) * bin_h / grid_h;
      for (int j = 0; j < grid_w; j++) {
        const float w = wstart + (j + .5f) * bin_w / grid_w;
        int ti, bi, li, ri;
        float v, u;
        _RoiAlignIntpParam(H, W, h, w, ti, bi, li, ri, v, u);
        if (li >= 0 && ri >= 0 && ti >= 0 && bi >= 0) {
          const float db = dyi * v;
          const float dt = dyi * (1.f - v);
          math::utils::AtomicAdd(offset_dx + ti * W + li, (1.f - u) * dt);
          math::utils::AtomicAdd(offset_dx + ti * W + ri, u * dt);
          math::utils::AtomicAdd(offset_dx + bi * W + li, (1.f - u) * db);
          math::utils::AtomicAdd(offset_dx + bi * W + ri, u * db);
        }
      } // End i
    } // End j
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

#define DEFINE_KERNEL_LAUNCHER(name, InputT, OutputT)                     \
  template <>                                                             \
  void name<InputT, CUDAContext>(                                         \
      const int C,                                                        \
      const int H,                                                        \
      const int W,                                                        \
      const int out_h,                                                    \
      const int out_w,                                                    \
      const int num_rois,                                                 \
      const float spatial_scale,                                          \
      const int sampling_ratio,                                           \
      const InputT* x,                                                    \
      const float* rois,                                                  \
      OutputT* y,                                                         \
      CUDAContext* ctx) {                                                 \
    auto nthreads = num_rois * C * out_h * out_w;                         \
    _##name<math::ScalarType<InputT>::type, float>                        \
        <<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
            nthreads,                                                     \
            C,                                                            \
            H,                                                            \
            W,                                                            \
            out_h,                                                        \
            out_w,                                                        \
            spatial_scale,                                                \
            sampling_ratio,                                               \
            reinterpret_cast<const math::ScalarType<InputT>::type*>(x),   \
            rois,                                                         \
            reinterpret_cast<math::ScalarType<OutputT>::type*>(y));       \
  }

DEFINE_KERNEL_LAUNCHER(RoiAlign, float16, float16);
DEFINE_KERNEL_LAUNCHER(RoiAlign, float, float);
DEFINE_KERNEL_LAUNCHER(RoiAlign, double, double);
DEFINE_KERNEL_LAUNCHER(RoiAlignGrad, float16, float); // RoiAlignGrad
DEFINE_KERNEL_LAUNCHER(RoiAlignGrad, float, float); // RoiAlignGrad
DEFINE_KERNEL_LAUNCHER(RoiAlignGrad, double, float); // RoiAlignGrad
#undef DEFINE_KERNEL_LAUNCHER

} // namespace kernels

} // namespace dragon

#endif // USE_CUDA
