
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernels {

namespace {

template <typename T>
__global__ void _Sigmoid(const int N, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    y[i] = T(1) / (T(1) + exp(-x[i]));
  }
}

template <>
__global__ void _Sigmoid<half>(const int N, const half* x, half* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    y[i] = __float2half(1.f / (1.f + exp(-__half2float(x[i]))));
  }
}

template <>
__global__ void _Sigmoid<half2>(const int N, const half2* x, half2* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const float2 val = __half22float2(x[i]);
    y[i] =
        __floats2half2_rn(1.f / (1.f + exp(-val.x)), 1.f / (1.f + exp(-val.y)));
  }
}

template <typename T>
__global__ void _SigmoidGrad(const int N, const T* dy, const T* y, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dx[i] = dy[i] * __ldg(y + i) * (1 - __ldg(y + i));
  }
}

template <>
__global__ void
_SigmoidGrad<half>(const int N, const half* dy, const half* y, half* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const float val = __half2float(y[i]);
    dx[i] = __float2half(__half2float(dy[i]) * val * (1.f - val));
  }
} // SigmoidGrad

template <>
__global__ void
_SigmoidGrad<half2>(const int N, const half2* dy, const half2* y, half2* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const float2 val = __half22float2(y[i]);
    const float2 grad = __half22float2(dy[i]);
    dx[i] = __floats2half2_rn(
        grad.x * val.x * (1.f - val.x), grad.y * val.y * (1.f - val.y));
  }
} // SigmoidGrad

} // namespace

/* ------------------- Launcher Separator ------------------- */

template <>
void Sigmoid<float16, CUDAContext>(
    const int N,
    const float16* x,
    float16* y,
    CUDAContext* ctx) {
  if ((N & 1) == 0) {
    _Sigmoid<<<CUDA_BLOCKS(N >> 1), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
        N >> 1, reinterpret_cast<const half2*>(x), reinterpret_cast<half2*>(y));
  } else {
    _Sigmoid<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
        N, reinterpret_cast<const half*>(x), reinterpret_cast<half*>(y));
  }
}

template <>
void SigmoidGrad<float16, CUDAContext>(
    const int N,
    const float16* dy,
    const float16* y,
    float16* dx,
    CUDAContext* ctx) {
  if ((N & 1) == 0) {
    _SigmoidGrad<<<CUDA_BLOCKS(N >> 1), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
        N >> 1,
        reinterpret_cast<const half2*>(dy),
        reinterpret_cast<const half2*>(y),
        reinterpret_cast<half2*>(dx));
  } else {
    _SigmoidGrad<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
        N,
        reinterpret_cast<const half*>(dy),
        reinterpret_cast<const half*>(y),
        reinterpret_cast<half*>(dx));
  }
} // SigmoidGrad

#define DEFINE_KERNEL_LAUNCHER(T)                                      \
  template <>                                                          \
  void Sigmoid<T, CUDAContext>(                                        \
      const int N, const T* x, T* y, CUDAContext* ctx) {               \
    _Sigmoid<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
        N, x, y);                                                      \
  }

#define DEFINE_GRAD_KERNEL_LAUNCHER(T)                                     \
  template <>                                                              \
  void SigmoidGrad<T, CUDAContext>(                                        \
      const int N, const T* dy, const T* y, T* dx, CUDAContext* ctx) {     \
    _SigmoidGrad<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
        N, dy, y, dx);                                                     \
  }

DEFINE_KERNEL_LAUNCHER(float);
DEFINE_KERNEL_LAUNCHER(double);
DEFINE_GRAD_KERNEL_LAUNCHER(float);
DEFINE_GRAD_KERNEL_LAUNCHER(double);
#undef DEFINE_KERNEL_LAUNCHER
#undef DEFINE_GRAD_KERNEL_LAUNCHER

} // namespace kernels

} // namespace dragon

#endif // USE_CUDA
