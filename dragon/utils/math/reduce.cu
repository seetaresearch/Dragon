
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/workspace.h"
#include "dragon/utils/device/common_cub.h"
#include "dragon/utils/device/common_thrust.h"
#include "dragon/utils/math/blas.h"
#include "dragon/utils/math/functional.h"
#include "dragon/utils/math/reduce.h"
#include "dragon/utils/math/utils.h"

namespace dragon {

namespace math {

namespace {

template <typename T, typename AccT, class Reducer>
__global__ void _RowwiseReduce(
    const int rows,
    const int cols,
    const Reducer reducer,
    const AccT init,
    const AccT scale,
    const T* x,
    T* y) {
  __shared__ typename BlockReduce<AccT>::TempStorage storage;
  CUDA_2D_KERNEL_LOOP1(i, cols) {
    AccT val = init;
    CUDA_2D_KERNEL_LOOP2(j, rows) {
      val = reducer(val, convert::To<AccT>(x[j * cols + i]));
    }
    val = BlockReduce<AccT>(storage).Reduce(val, reducer);
    if (threadIdx.x == 0) {
      y[i] = convert::To<T>(val * scale);
    }
  }
}

template <typename T, typename AccT, class Reducer>
__global__ void _ColwiseReduce(
    const int rows,
    const int cols,
    const Reducer reducer,
    const AccT init,
    const AccT scale,
    const T* x,
    T* y) {
  __shared__ typename BlockReduce<AccT>::TempStorage storage;
  CUDA_2D_KERNEL_LOOP1(i, rows) {
    AccT val = init;
    CUDA_2D_KERNEL_LOOP2(j, cols) {
      val = reducer(val, convert::To<AccT>(x[i * cols + j]));
    }
    val = BlockReduce<AccT>(storage).Reduce(val, reducer);
    if (threadIdx.x == 0) {
      y[i] = convert::To<T>(val * scale);
    }
  }
}

template <typename T, typename AccT, class Reducer, int D>
__global__ void _GenericReduce(
    const int rows,
    const int cols,
    const SimpleArray<int, D> x_dims,
    const SimpleArray<int, D> x_strides,
    const Reducer reducer,
    const AccT init,
    const AccT scale,
    const T* x,
    T* y) {
  __shared__ typename BlockReduce<AccT>::TempStorage storage;
  CUDA_2D_KERNEL_LOOP1(i, rows) {
    AccT val = init;
    CUDA_2D_KERNEL_LOOP2(j, cols) {
      int xi = 0, c = i * cols + j;
#pragma unroll
      for (int d = D - 1; d >= 0; --d) {
        int r;
        FIXED_DIVISOR_DIV_MOD(x_dims.data[d], c, &c, &r);
        xi += r * x_strides.data[d];
      }
      val = reducer(val, convert::To<AccT>(x[xi]));
    }
    val = BlockReduce<AccT>(storage).Reduce(val, reducer);
    if (threadIdx.x == 0) {
      y[i] = convert::To<T>(val * scale);
    }
  }
}

template <typename T, typename AccT, class Reducer, int D>
void _GenericReduceImpl(
    const int* dims,
    const int num_axes,
    const int* axes,
    const Reducer reducer,
    const AccT init,
    const AccT scale,
    const T* x,
    T* y,
    CUDAContext* ctx) {
  SimpleArray<int, D> transpose_axes;
  SimpleArray<int, D> transpose_strides;
  SimpleArray<int, D> transpose_dims;
  math::utils::TransposeAxesForReduce(D, num_axes, axes, transpose_axes.data);
  math::utils::ComputeTransposeStrides(
      D, dims, transpose_axes.data, transpose_strides.data);
  int rows = 1, cols = 1;
  const int pivot = D - num_axes;
  for (int i = 0; i < pivot; ++i) {
    rows *= dims[transpose_axes.data[i]];
  }
  for (int i = pivot; i < D; ++i) {
    cols *= dims[transpose_axes.data[i]];
  }
  for (int i = 0; i < D; ++i) {
    transpose_dims.data[i] = dims[transpose_axes.data[i]];
  }
  _GenericReduce<<<rows, CUDA_THREADS, 0, ctx->cuda_stream()>>>(
      rows,
      cols,
      transpose_dims,
      transpose_strides,
      reducer,
      init,
      scale,
      x,
      y);
}

#define DEFINE_REDUCE_DISPATCHER(name)                               \
  template <typename T, typename AccT, typename Reducer>             \
  void _Reduce##name(                                                \
      const int num_dims,                                            \
      const int* dims,                                               \
      const int num_axes,                                            \
      const int* axes,                                               \
      const Reducer reducer,                                         \
      const AccT init,                                               \
      const AccT scale,                                              \
      const T* x,                                                    \
      T* y,                                                          \
      CUDAContext* ctx) {                                            \
    int rows, cols;                                                  \
    vec32_t out_dims(dims, dims + num_dims);                         \
    for (int i = 0; i < num_axes; ++i) {                             \
      out_dims[axes[i]] = 1;                                         \
    }                                                                \
    if (math::utils::IsRowwiseReduce(                                \
            num_dims, dims, out_dims.data(), &rows, &cols)) {        \
      _RowwiseReduce<<<cols, CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
          rows, cols, reducer, init, scale, x, y);                   \
      return;                                                        \
    }                                                                \
    if (math::utils::IsColwiseReduce(                                \
            num_dims, dims, out_dims.data(), &rows, &cols)) {        \
      _ColwiseReduce<<<rows, CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
          rows, cols, reducer, init, scale, x, y);                   \
      return;                                                        \
    }                                                                \
    CUDA_TENSOR_DIMS_CHECK(num_dims);                                \
    DISPATCH_FUNC_BY_VALUE_WITH_TYPE_3(                              \
        _GenericReduceImpl,                                          \
        T,                                                           \
        AccT,                                                        \
        Reducer,                                                     \
        num_dims,                                                    \
        dims,                                                        \
        num_axes,                                                    \
        axes,                                                        \
        reducer,                                                     \
        init,                                                        \
        scale,                                                       \
        x,                                                           \
        y,                                                           \
        ctx);                                                        \
  }

DEFINE_REDUCE_DISPATCHER(Max);
DEFINE_REDUCE_DISPATCHER(Min);
DEFINE_REDUCE_DISPATCHER(Sum);
#undef DEFINE_REDUCE_DISPATCHER

} // namespace

// Disable FP16 DeviceReduce.
// We found that FP16 accumulator drops too many small values in
// empirical experiments.
template <>
DRAGON_API void ReduceSum<float16, CUDAContext>(
    const int num_dims,
    const int* dims,
    const int num_axes,
    const int* axes,
    const float scale,
    const float16* x,
    float16* y,
    CUDAContext* ctx) {
  // NB: Performance may drop in some cases.
  _ReduceSum(
      num_dims,
      dims,
      num_axes,
      axes,
      math::PlusFunctor<float>(),
      0.f,
      scale,
      x,
      y,
      ctx);
}

#define DEFINE_REDUCE_FUNC(name, T, AccT, Reducer, kInit)                  \
  template <>                                                              \
  DRAGON_API void Reduce##name<T, CUDAContext>(                            \
      const int num_dims,                                                  \
      const int* dims,                                                     \
      const int num_axes,                                                  \
      const int* axes,                                                     \
      const float scale,                                                   \
      const T* x,                                                          \
      T* y,                                                                \
      CUDAContext* ctx) {                                                  \
    const int count =                                                      \
        std::accumulate(dims, dims + num_dims, 1, std::multiplies<int>()); \
    if (num_dims == num_axes && count > 10000) {                           \
      size_t ws_nbytes = 0;                                                \
      cub::DeviceReduce::Reduce(                                           \
          nullptr,                                                         \
          ws_nbytes,                                                       \
          x,                                                               \
          y,                                                               \
          count,                                                           \
          Reducer<T>(),                                                    \
          convert::To<T>(kInit),                                           \
          ctx->cuda_stream());                                             \
      cub::DeviceReduce::Reduce(                                           \
          ctx->workspace()->data<CUDAContext>({ws_nbytes}, "data:1")[0],   \
          ws_nbytes,                                                       \
          x,                                                               \
          y,                                                               \
          count,                                                           \
          Reducer<T>(),                                                    \
          convert::To<T>(kInit),                                           \
          ctx->cuda_stream());                                             \
      math::Scale(1, scale, y, y, ctx);                                    \
      return;                                                              \
    }                                                                      \
    _Reduce##name(                                                         \
        num_dims,                                                          \
        dims,                                                              \
        num_axes,                                                          \
        axes,                                                              \
        Reducer<AccT>(),                                                   \
        convert::To<AccT>(kInit),                                          \
        convert::To<AccT>(scale),                                          \
        x,                                                                 \
        y,                                                                 \
        ctx);                                                              \
  }

DEFINE_REDUCE_FUNC(
    Max,
    uint8_t,
    uint8_t,
    math::MaxFunctor,
    std::numeric_limits<uint8_t>::lowest());
DEFINE_REDUCE_FUNC(
    Max,
    int8_t,
    int8_t,
    math::MaxFunctor,
    std::numeric_limits<int8_t>::lowest());
DEFINE_REDUCE_FUNC(
    Max,
    int,
    int,
    math::MaxFunctor,
    std::numeric_limits<int>::lowest());
DEFINE_REDUCE_FUNC(
    Max,
    int64_t,
    int64_t,
    math::MaxFunctor,
    std::numeric_limits<int64_t>::lowest());
DEFINE_REDUCE_FUNC(
    Max,
    float16,
    float,
    math::MaxFunctor,
    cub::Traits<half>::Lowest());
DEFINE_REDUCE_FUNC(
    Max,
    float,
    float,
    math::MaxFunctor,
    std::numeric_limits<float>::lowest());
DEFINE_REDUCE_FUNC(
    Max,
    double,
    double,
    math::MaxFunctor,
    std::numeric_limits<double>::lowest());
DEFINE_REDUCE_FUNC(
    Min,
    uint8_t,
    uint8_t,
    math::MinFunctor,
    std::numeric_limits<uint8_t>::max());
DEFINE_REDUCE_FUNC(
    Min,
    int8_t,
    int8_t,
    math::MinFunctor,
    std::numeric_limits<int8_t>::max());
DEFINE_REDUCE_FUNC(
    Min,
    int,
    int,
    math::MinFunctor,
    std::numeric_limits<int>::max());
DEFINE_REDUCE_FUNC(
    Min,
    int64_t,
    int64_t,
    math::MinFunctor,
    std::numeric_limits<int64_t>::max());
DEFINE_REDUCE_FUNC(
    Min,
    float16,
    float,
    math::MinFunctor,
    cub::Traits<half>::Max());
DEFINE_REDUCE_FUNC(
    Min,
    float,
    float,
    math::MinFunctor,
    std::numeric_limits<float>::max());
DEFINE_REDUCE_FUNC(
    Min,
    double,
    double,
    math::MinFunctor,
    std::numeric_limits<double>::max());
DEFINE_REDUCE_FUNC(Sum, uint8_t, uint8_t, math::PlusFunctor, uint8_t(0));
DEFINE_REDUCE_FUNC(Sum, int8_t, int8_t, math::PlusFunctor, int8_t(0));
DEFINE_REDUCE_FUNC(Sum, int, int, math::PlusFunctor, int(0));
DEFINE_REDUCE_FUNC(Sum, int64_t, int64_t, math::PlusFunctor, int64_t(0));
DEFINE_REDUCE_FUNC(Sum, float, float, math::PlusFunctor, 0.f);
DEFINE_REDUCE_FUNC(Sum, double, double, math::PlusFunctor, 0.);
#undef DEFINE_REDUCE_FUNC

#define DEFINE_SUM_FUNC(T)                                                  \
  template <>                                                               \
  DRAGON_API void Sum<T, CUDAContext>(                                      \
      const int N, const float alpha, const T* x, T* y, CUDAContext* ctx) { \
    vec32_t dims = {N}, axes = {0};                                         \
    math::ReduceSum(1, dims.data(), 1, axes.data(), alpha, x, y, ctx);      \
  }

DEFINE_SUM_FUNC(uint8_t);
DEFINE_SUM_FUNC(int8_t);
DEFINE_SUM_FUNC(int);
DEFINE_SUM_FUNC(int64_t);
DEFINE_SUM_FUNC(float16);
DEFINE_SUM_FUNC(float);
DEFINE_SUM_FUNC(double);
#undef DEFINE_SUM_FUNC

#define DEFINE_SUM_FUNC(T)                                            \
  template <>                                                         \
  DRAGON_API T Sum<T, CUDAContext>(                                   \
      const int N, const float alpha, const T* x, CUDAContext* ctx) { \
    auto policy = thrust::cuda::par.on(ctx->cuda_stream());           \
    auto val = thrust::reduce(policy, x, x + N) * alpha;              \
    return static_cast<T>(val);                                       \
  }

DEFINE_SUM_FUNC(uint8_t);
DEFINE_SUM_FUNC(int8_t);
DEFINE_SUM_FUNC(int);
DEFINE_SUM_FUNC(int64_t);
DEFINE_SUM_FUNC(float);
DEFINE_SUM_FUNC(double);
#undef DEFINE_SUM_FUNC

} // namespace math

} // namespace dragon

#endif // USE_CUDA
